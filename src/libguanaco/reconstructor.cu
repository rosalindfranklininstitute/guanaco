#include "hip/hip_runtime.h"
/*
 * This file is part of guanaco-ctf.
 * Copyright 2021 Diamond Light Source
 * Copyright 2021 Rosalind Franklin Institute
 *
 * Author: James Parkhurst
 *
 * guanaco-ctf is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * guanaco-ctf is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with guanaco-ctf. If not, see <http:// www.gnu.org/licenses/>.
 */
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <guanaco/guanaco.h>

namespace guanaco {

namespace detail {

  // Declare global variables
  namespace global {

    const size_t MAX_ANGLES = 4096;

    __constant__ float angle_sin[MAX_ANGLES];
    __constant__ float angle_cos[MAX_ANGLES];
    __constant__ float angle_offset[MAX_ANGLES];
    __constant__ float angle_scale[MAX_ANGLES];

  }  // namespace global

  namespace g = global;

  /**
   * A functor that implements the back projection per voxel
   */
  struct BPFunction {
    hipTextureObject_t sinogram;
    size_t num_angles;
    size_t grid_width;
    size_t grid_height;
    float output_scale;
    float dscale;
    float doffset;

    BPFunction(hipTextureObject_t sinogram_, 
               size_t num_angles_,
               size_t grid_width_,
               size_t grid_height_,
               float output_scale_,
               float dscale_,
               float doffset_)
        : sinogram(sinogram_),
          num_angles(num_angles_),
          grid_width(grid_width_),
          grid_height(grid_height_),
          output_scale(output_scale_),
          dscale(dscale_),
          doffset(doffset_) {
      GUANACO_ASSERT(num_angles_ <= g::MAX_ANGLES);
    }

    __device__ float operator()(size_t index, float voxel) const {
      // Get the X and Y indices
      const size_t j = index / grid_width;
      const size_t i = index - j * grid_width;

      // Compute the x and y coordinates
      const float x = (i - 0.5 * grid_width + 0.5);
      const float y = (j - 0.5 * grid_height + 0.5);

      // Loop through all the angles and compute the value of the voxel
      float value = 0.0f;
      for (size_t angle = 0; angle < num_angles; ++angle) {
        // Get parameters
        const float cos_angle = g::angle_cos[angle];
        const float sin_angle = g::angle_sin[angle];
        const float offset = g::angle_offset[angle];
        const float scale = g::angle_scale[angle];

        // Compute the pixel and defocus coordinate
        const float pixel = cos_angle * x - sin_angle * y + offset;
        const float height = -sin_angle * x - cos_angle * y;
        const float defocus = height * dscale + doffset;

        // Sum the sinogram value for the pixel and angle
        value += tex3D<float>(sinogram, pixel, angle + 0.5, defocus) * scale;
      }

      // Add the contribution to the voxel
      return voxel + value * output_scale;
    }
  };

  struct BP {
    using size_type = std::size_t;

    hipTextureObject_t sinogram_;
    hipArray *sinogram_array_;
    size_type num_pixels_;
    size_type num_angles_;
    size_type num_defocus_;
    float pixel_size_;
    float min_defocus_;
    float max_defocus_;

    BP(size_type num_pixels,
       size_type num_angles,
       size_type num_defocus,
       float centre,
       float pixel_size,
       float min_defocus,
       float max_defocus,
       const float *sinogram,
       const float *angles)
        : sinogram_(0),
          sinogram_array_(nullptr),
          num_pixels_(num_pixels),
          num_angles_(num_angles),
          num_defocus_(num_defocus),
          pixel_size_(pixel_size),
          min_defocus_(min_defocus),
          max_defocus_(max_defocus) {
      // Check input
      GUANACO_ASSERT(num_pixels_ > 0);
      GUANACO_ASSERT(num_angles_ > 0);
      GUANACO_ASSERT(num_defocus_ > 0);
      GUANACO_ASSERT(pixel_size_ > 0);
      GUANACO_ASSERT(max_defocus_ >= min_defocus_);

      // Copy the angle data to device symbols
      copy_angles(angles, num_angles_, centre);

      // Copy the sinogram to the texture memory
      copy_sinogram(sinogram, num_pixels, num_angles, num_defocus);
    }

    ~BP() {
      hipDestroyTextureObject(sinogram_);
      hipFreeArray(sinogram_array_);
    }

    void copy_angles(const float *angles,
                     size_type num_angles,
                     float centre) const {
      // Copy the data to the symbol. For some reason I can't pass the symbol
      // pointer as normal (no idea) so I have to pass a pointer to the
      // symbol array pointer and then dereference!
      auto copy = [](auto symbol, auto data, auto n) {
        GUANACO_ASSERT(n <= g::MAX_ANGLES);
        auto error = hipMemcpyToSymbol(HIP_SYMBOL(
          *symbol), data, n * sizeof(float), 0, hipMemcpyHostToDevice);
        GUANACO_ASSERT_CUDA(error == hipSuccess);
      };

      // Allocate host vectors
      auto angle_sin = thrust::host_vector<float>(num_angles);
      auto angle_cos = thrust::host_vector<float>(num_angles);
      auto angle_offset = thrust::host_vector<float>(num_angles);
      auto angle_scale = thrust::host_vector<float>(num_angles);

      // Compute the quanities to store in the symbols
      for (auto i = 0; i < num_angles; ++i) {
        // Get the ray vector and length of the ray vector
        auto angle = angles[i];
        auto cos_angle = std::cos(angle);
        auto sin_angle = std::sin(angle);
        auto det_x0 = -centre * cos_angle;
        auto det_y0 = -centre * sin_angle;
        auto ray_length =
          std::sqrt(cos_angle * cos_angle + sin_angle * sin_angle);
        auto d = cos_angle * cos_angle + sin_angle * sin_angle;

        // Fill the arrays
        angle_cos[i] = cos_angle / d;
        angle_sin[i] = sin_angle / d;
        angle_offset[i] = (det_y0 * (-sin_angle) - det_x0 * cos_angle) / d;
        angle_scale[i] = ray_length / std::abs(d);
      }

      // Copy the arrays to the symbols
      copy(&g::angle_sin, angle_sin.data(), num_angles);
      copy(&g::angle_cos, angle_cos.data(), num_angles);
      copy(&g::angle_offset, angle_offset.data(), num_angles);
      copy(&g::angle_scale, angle_scale.data(), num_angles);
    }

    void copy_sinogram(const float *sinogram,
                       size_type num_pixels,
                       size_type num_angles,
                       size_type num_defocus) {
      // Allocate a cuda array needed to bind 3D texture
      auto channel_desc = hipCreateChannelDesc<float>();
      auto extent = make_hipExtent(num_pixels, num_angles, num_defocus);
      auto error = hipMalloc3DArray(&sinogram_array_, &channel_desc, extent);
      GUANACO_ASSERT_CUDA(error == hipSuccess);

      // Copy the data
      hipMemcpy3DParms copy_params{0};
      copy_params.srcPtr = make_hipPitchedPtr((void *)sinogram,
                                               extent.width * sizeof(float),
                                               extent.width,
                                               extent.height);
      copy_params.dstArray = sinogram_array_;
      copy_params.extent = extent;
      copy_params.kind = hipMemcpyDeviceToDevice;
      error = hipMemcpy3D(&copy_params);
      GUANACO_ASSERT_CUDA(error == hipSuccess);

      // create texture object
      hipResourceDesc resDesc;
      memset(&resDesc, 0, sizeof(resDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array = sinogram_array_;

      // Set the texture parameters
      // For pixels outside the expected range, this sets the
      // value to zero (border). For defocus outside of expected range, use the
      // closest (clamp).
      hipTextureDesc texDesc;
      memset(&texDesc, 0, sizeof(texDesc));
      texDesc.readMode = hipReadModeElementType;
      texDesc.addressMode[0] = hipAddressModeBorder;
      texDesc.addressMode[1] = hipAddressModeBorder;
      texDesc.addressMode[2] = hipAddressModeClamp;
      texDesc.filterMode = hipFilterModeLinear;
      texDesc.normalizedCoords = false;

      // create texture object: we only have to do this once!
      error = hipCreateTextureObject(&sinogram_, &resDesc, &texDesc, NULL);
      GUANACO_ASSERT_CUDA(error == hipSuccess);
    }

    void launch(float *reconstruction,
                size_type grid_width,
                size_type grid_height,
                float scale) const {
      // Check the input
      GUANACO_ASSERT(num_defocus_ == 1 || max_defocus_ > min_defocus_);

      // Compute the defocus scale and offset
      auto dscale = num_defocus_ > 1 ? num_defocus_ * pixel_size_
                                         / (max_defocus_ - min_defocus_)
                                     : 0;
      auto doffset = -dscale * (min_defocus_ / pixel_size_);

      // Get some other quantities
      auto grid_size = grid_width * grid_height;
      auto index = thrust::counting_iterator<size_t>(0);
      auto recon = thrust::device_pointer_cast(reconstruction);
      
      // Initialise the functor
      BPFunction func(
        sinogram_, 
        num_angles_, 
        grid_width, 
        grid_height, 
        scale, 
        dscale, 
        doffset);

      // Do the reconstruction
      thrust::transform(index, index + grid_size, recon, recon, func);
    }
  };

}  // namespace detail

template <>
Reconstructor_t<e_device>::Reconstructor_t(const Config &config)
    : config_(config),
      filter_(config_.num_pixels, config_.num_angles, config_.num_defocus) {
  GUANACO_ASSERT(config_.device == e_device);
  GUANACO_ASSERT(config_.is_valid());
}

template <>
void Reconstructor_t<e_device>::project(const float *sinogram,
                                        float *reconstruction) const {
  // Check the dimensions against the maximum texture size
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  GUANACO_ASSERT(prop.maxTexture3D[1] >= detail::g::MAX_ANGLES);
  GUANACO_ASSERT(prop.maxTexture3D[0] >= config_.num_pixels);
  GUANACO_ASSERT(prop.maxTexture3D[1] >= config_.num_angles);
  GUANACO_ASSERT(prop.maxTexture3D[2] >= config_.num_defocus);

  // Compute the scale
  auto scale = M_PI / (2 * config_.num_angles);

  // Initialise the back projector class
  auto bp = detail::BP(config_.num_pixels,
                       config_.num_angles,
                       config_.num_defocus,
                       config_.centre,
                       config_.pixel_size,
                       config_.min_defocus,
                       config_.max_defocus,
                       sinogram,
                       config_.angles.data());

  // Launch the back projector
  bp.launch(reconstruction, config_.grid_width, config_.grid_height, scale);
}

template <>
void Reconstructor_t<e_device>::operator()(const float *sinogram,
                                           float *reconstruction) const {
  // A function to set the gpu index
  auto set_gpu_index = [](int index) {
    if (index >= 0) {
      hipSetDevice(index);
      auto error = hipGetLastError();
      GUANACO_ASSERT_CUDA((error == hipSuccess)
                          || (error == hipErrorSetOnActiveProcess));
    }
  };

  // Make some typedefs
  using vector_type = thrust::device_vector<float>;

  // Get the sinogram and reconstruction sizes along with the number of
  // angles and the pixel area
  auto sino_size = config_.sino_size();
  auto grid_size = config_.grid_size();

  // Allocate device vectors for sinogram and reconstruction
  auto sinogram_d = vector_type(sinogram, sinogram + sino_size);
  auto reconstruction_d = vector_type(grid_size, 0);

  // Set the gpu
  set_gpu_index(config_.gpu_index);

  // Filter the sinogram
  filter_(sinogram_d.data().get());

  // Perform the backprojection
  project(sinogram_d.data().get(), reconstruction_d.data().get());

  // Copy the data back to the host ptr
  thrust::copy(
    reconstruction_d.begin(), reconstruction_d.end(), reconstruction);
}

}  // namespace guanaco
